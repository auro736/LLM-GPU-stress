// gpu_stress_test.cu

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>


#include <iostream>
#include <chrono>
#include <thread>

// Define constants
const int MATRIX_SIZE = 1024;
const int NUM_THREADS = 256;
const int NUM_BLOCKS = 256;
const int TEST_DURATION = 60; // seconds

// Kernel function to stress computational units
__global__ void stress_compute(float *matrix1, float *matrix2, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float sum = 0.0f;
        for (int i = 0; i < MATRIX_SIZE; i++) {
            sum += matrix1[idx * MATRIX_SIZE + i] * matrix2[i * MATRIX_SIZE + idx % MATRIX_SIZE];
        }
        result[idx] = sum;
    }
}

// Kernel function to stress memory
__global__ void stress_memory(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        array[idx] = array[idx] * 2.0f;
    }
}

// Kernel function to stress XU units
__global__ void stress_xu(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        array[idx] = __sinf(array[idx]);
    }
}

// Kernel function to stress atomic operations
__global__ void stress_atomic(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        atomicAdd(&array[idx], 1.0f);
    }
}

int main() {
    // Allocate host memory
    float *h_matrix1, *h_matrix2, *h_result, *h_array;
    hipHostMalloc((void **)&h_matrix1, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_matrix2, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_result, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_array, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);

    // Initialize host memory
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        h_matrix1[i] = 1.0f;
        h_matrix2[i] = 2.0f;
        h_result[i] = 0.0f;
        h_array[i] = 1.0f;
    }

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_result, *d_array;
    hipMalloc((void **)&d_matrix1, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_matrix2, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_result, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_array, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));

    // Copy host memory to device memory
    hipMemcpy(d_matrix1, h_matrix1, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, h_matrix2, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_array, h_array, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    auto start_time = std::chrono::high_resolution_clock::now();
    while (true) {
        stress_compute<<<NUM_BLOCKS, NUM_THREADS>>>(d_matrix1, d_matrix2, d_result);
        stress_memory<<<NUM_BLOCKS, NUM_THREADS>>>(d_array);
        stress_xu<<<NUM_BLOCKS, NUM_THREADS>>>(d_array);
        stress_atomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_array);
        hipDeviceSynchronize();
        auto current_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::seconds>(current_time - start_time).count();
        if (duration > TEST_DURATION) {
            break;
        }
    }

    // Free host and device memory
    hipHostFree(h_matrix1);
    hipHostFree(h_matrix2);
    hipHostFree(h_result);
    hipHostFree(h_array);
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);
    hipFree(d_array);

    return 0;
}