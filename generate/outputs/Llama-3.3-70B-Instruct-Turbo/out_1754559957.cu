// gpu_stress_test.cu

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>


#include <cmath>
#include <iostream>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
#define MATRIX_SIZE 1024
#define TEST_DURATION 60 // seconds

__global__ void matrixMultiplicationKernel(float *A, float *B, float *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < MATRIX_SIZE && idy < MATRIX_SIZE) {
        float sum = 0.0f;
        for (int i = 0; i < MATRIX_SIZE; i++) {
            sum += A[idy * MATRIX_SIZE + i] * B[i * MATRIX_SIZE + idx];
        }
        C[idy * MATRIX_SIZE + idx] = sum;
    }
}

__global__ void floatingPointCalculationKernel(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float value = array[idx];
        for (int i = 0; i < 1000; i++) {
            value = sqrtf(value) * cosf(value) * sinf(value);
        }
        array[idx] = value;
    }
}

__global__ void atomicOperationKernel(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float value = 1.0f;
        for (int i = 0; i < 1000; i++) {
            atomicAdd(&array[idx], value);
        }
    }
}

__global__ void specialFunctionKernel(float *array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float value = array[idx];
        for (int i = 0; i < 1000; i++) {
            value = __sinf(value) * __cosf(value) * __expf(value);
        }
        array[idx] = value;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <test_duration_in_seconds>" << std::endl;
        return 1;
    }

    int testDuration = std::stoi(argv[1]);

    float *h_A, *h_B, *h_C, *h_array;
    float *d_A, *d_B, *d_C, *d_array;

    size_t matrixSize = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);
    size_t arraySize = MATRIX_SIZE * MATRIX_SIZE * sizeof(float);

    hipHostMalloc((void **)&h_A, matrixSize, hipHostMallocDefault);
    hipHostMalloc((void **)&h_B, matrixSize, hipHostMallocDefault);
    hipHostMalloc((void **)&h_C, matrixSize, hipHostMallocDefault);
    hipHostMalloc((void **)&h_array, arraySize, hipHostMallocDefault);

    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, matrixSize);
    hipMalloc((void **)&d_C, matrixSize);
    hipMalloc((void **)&d_array, arraySize);

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX;
        h_array[i] = (float)rand() / RAND_MAX;
    }

    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_array, h_array, arraySize, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    while (true) {
        matrixMultiplicationKernel<<<dim3(GRID_SIZE, GRID_SIZE), dim3(BLOCK_SIZE, BLOCK_SIZE)>>>(d_A, d_B, d_C);
        floatingPointCalculationKernel<<<dim3(GRID_SIZE), dim3(BLOCK_SIZE)>>>(d_array);
        atomicOperationKernel<<<dim3(GRID_SIZE), dim3(BLOCK_SIZE)>>>(d_array);
        specialFunctionKernel<<<dim3(GRID_SIZE), dim3(BLOCK_SIZE)>>>(d_array);

        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        if (milliseconds / 1000.0f > testDuration) {
            break;
        }
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_array);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_array);

    return 0;
}