// cuda_stress_test.cu

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>


#include <cmath>
#include <iostream>

// Define constants
#define BLOCK_SIZE 256
#define GRID_SIZE 256
#define MATRIX_SIZE 1024
#define TEST_DURATION 60 // seconds

// Define kernel for matrix multiplication
__global__ void matrixMultKernel(float *A, float *B, float *C, int size) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < size && col < size) {
        float sum = 0.0f;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}

// Define kernel for floating-point calculations
__global__ void floatCalcKernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx];
        val = sinf(val) + cosf(val) + expf(val) + logf(val);
        data[idx] = val;
    }
}

// Define kernel for special functions stressing XU units
__global__ void specialFuncKernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx];
        val = sqrtf(val) + rsqrtf(val) + rcbrtf(val);
        data[idx] = val;
    }
}

// Define kernel for atomic operations
__global__ void atomicOpKernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx];
        atomicAdd(&data[idx], val);
    }
}

// Define kernel for memory access pattern
__global__ void memoryAccessKernel(float *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = data[idx];
        data[idx + size / 2] = val;
        __syncthreads();
        val = data[idx + size / 2];
        data[idx] = val;
    }
}

int main() {
    // Initialize CUDA device
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }
    hipSetDevice(0);

    // Allocate host memory
    float *h_A, *h_B, *h_C, *h_data;
    hipHostMalloc((void **)&h_A, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_B, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_C, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **)&h_data, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipHostMallocDefault);

    // Initialize host data
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        h_A[i] = (float)i;
        h_B[i] = (float)i;
        h_data[i] = (float)i;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_data;
    hipMalloc((void **)&d_A, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_B, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_C, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&d_data, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));

    // Transfer data from host to device
    hipMemcpy(d_A, h_A, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data, h_data, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernels
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(GRID_SIZE, GRID_SIZE);
    for (int i = 0; i < TEST_DURATION; i++) {
        matrixMultKernel<<<grid, block>>>(d_A, d_B, d_C, MATRIX_SIZE);
        floatCalcKernel<<<grid, block>>>(d_data, MATRIX_SIZE * MATRIX_SIZE);
        specialFuncKernel<<<grid, block>>>(d_data, MATRIX_SIZE * MATRIX_SIZE);
        atomicOpKernel<<<grid, block>>>(d_data, MATRIX_SIZE * MATRIX_SIZE);
        memoryAccessKernel<<<grid, block>>>(d_data, MATRIX_SIZE * MATRIX_SIZE);
        hipDeviceSynchronize();
    }

    // Transfer data from device to host
    hipMemcpy(h_C, d_C, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_data, d_data, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_data);

    // Free host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_data);

    return 0;
}