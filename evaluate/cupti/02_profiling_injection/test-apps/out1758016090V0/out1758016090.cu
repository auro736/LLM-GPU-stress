// stress_test.cu

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MATRIX_SIZE 1024
#define TEST_DURATION 60 // seconds
#define NUM_STREAMS 16

// Kernel to stress computational units
__global__ void compute_stress(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float result = 0.0f;
        for (int i = 0; i < 1000; i++) {
            result += sinf(data[idx] * i) * cosf(data[idx] * i);
        }
        data[idx] = result;
    }
}

// Kernel to stress memory
__global__ void memory_stress(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float result = 0.0f;
        for (int i = 0; i < 1000; i++) {
            result += data[(idx + i) % (MATRIX_SIZE * MATRIX_SIZE)];
        }
        data[idx] = result;
    }
}

// Kernel to stress XU units
__global__ void xu_stress(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float result = 0.0f;
        for (int i = 0; i < 1000; i++) {
            result += __expf(data[idx] * i) * __logf(data[idx] * i);
        }
        data[idx] = result;
    }
}

// Kernel to stress atomic operations
__global__ void atomic_stress(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MATRIX_SIZE * MATRIX_SIZE) {
        float result = 0.0f;
        for (int i = 0; i < 1000; i++) {
            atomicAdd(&data[idx], 1.0f);
        }
    }
}

int main() {
    // Initialize CUDA
    hipDeviceReset();
    int device;
    hipGetDevice(&device);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory
    float *data;
    hipMalloc((void **)&data, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));

    // Initialize data
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; i++) {
        data[i] = (float)rand() / RAND_MAX;
    }

    // Launch kernels
    dim3 blockSize(256);
    dim3 gridSize((MATRIX_SIZE * MATRIX_SIZE + blockSize.x - 1) / blockSize.x);
    clock_t start = clock();
    while ((clock() - start) / (double)CLOCKS_PER_SEC < TEST_DURATION) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            compute_stress<<<gridSize, blockSize, 0, streams[i]>>>(data);
            memory_stress<<<gridSize, blockSize, 0, streams[i]>>>(data);
            xu_stress<<<gridSize, blockSize, 0, streams[i]>>>(data);
            atomic_stress<<<gridSize, blockSize, 0, streams[i]>>>(data);
        }
    }

    // Clean up
    hipDeviceSynchronize();
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
    hipFree(data);
    hipDeviceReset();

    return 0;
}